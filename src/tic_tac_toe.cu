#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>

#define ROWS 3
#define COLS 3

bool checkWinHost(const int *board, int player) {

    for (int row = 0; row < ROWS; row++) {
        if (board[row*COLS] == player &&
            board[row*COLS+1] == player &&
            board[row*COLS+2] == player) {
            return true;
        }
    }

    for (int col = 0; col < COLS; col++) {
        if (board[col] == player &&
            board[col+COLS] == player &&
            board[col+2*COLS] == player) {
            return true;
        }
    }

    if (board[0] == player && board[4] == player && board[8] == player) {
        return true;
    }
    if (board[2] == player && board[4] == player && board[6] == player) {
        return true;
    }

    return false;
}

__device__ bool checkWinDevice(const int *board, int player) {
    for (int row = 0; row < ROWS; row++) {
        if (board[row*COLS] == player &&
            board[row*COLS+1] == player &&
            board[row*COLS+2] == player) {
            return true;
        }
    }
    for (int col = 0; col < COLS; col++) {
        if (board[col] == player &&
            board[col+COLS] == player &&
            board[col+2*COLS] == player) {
            return true;
        }
    }
    if (board[0] == player && board[4] == player && board[8] == player) {
        return true;
    }
    if (board[2] == player && board[4] == player && board[6] == player) {
        return true;
    }
    return false;
}

__global__ void randomMove(int *board, int player, unsigned long long seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx == 0) {
        hiprandState_t state;
        hiprand_init(seed, idx, 0, &state);

        while (true) {
            int pos = hiprand(&state) % (ROWS * COLS);
            if (board[pos] == 0) {
                board[pos] = player;
                return;
            }
        }
    }
}

__global__ void smartMove(int *board, int player) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx == 0) {
        int opponent = 3 - player;

        for (int i = 0; i < ROWS*COLS; i++) {
            if (board[i] == 0) {
                board[i] = player;
                if (checkWinDevice(board, player)) return;
                board[i] = 0;
            }
        }

        for (int i = 0; i < ROWS*COLS; i++) {
            if (board[i] == 0) {
                board[i] = opponent;
                if (checkWinDevice(board, opponent)) {
                    board[i] = player;
                    return;
                }
                board[i] = 0;
            }
        }

        if (board[4] == 0) {
            board[4] = player;
            return;
        }

        int corners[] = {0, 2, 6, 8};
        for (int i = 0; i < 4; i++) {
            if (board[corners[i]] == 0) {
                board[corners[i]] = player;
                return;
            }
        }

        for (int i = 0; i < ROWS*COLS; i++) {
            if (board[i] == 0) {
                board[i] = player;
                return;
            }
        }
    }
}

void printBoard(const int *board) {
    printf("\n");
    printf("    1   2   3\n");
    printf("  +---+---+---+\n");

    for (int row = 0; row < ROWS; row++) {
        printf("%d |", row+1);
        for (int col = 0; col < COLS; col++) {
            int val = board[row*COLS + col];
            printf(" %c |", val == 1 ? 'X' : val == 2 ? 'O' : ' ');
        }
        printf("\n");
        printf("  +---+---+---+\n");
    }
    printf("\n");
}

bool isBoardFull(const int *board) {
    for (int i = 0; i < ROWS*COLS; i++) {
        if (board[i] == 0) return false;
    }
    return true;
}

int main() {
    printf("Tic-Tac-Toe Game (CUDA)\n");
    printf("Player 1: X (Random moves)\n");
    printf("Player 2: O (Smart moves)\n\n");

    int *d_board;
    hipMalloc(&d_board, ROWS * COLS * sizeof(int));
    hipMemset(d_board, 0, ROWS * COLS * sizeof(int));

    int *h_board = (int *)malloc(ROWS * COLS * sizeof(int));

    int round = 1;
    int currentPlayer = 1; 

    while (round <= 9) {
        printf("Round %d - Player %d's turn (%c)\n",
               round, currentPlayer, currentPlayer == 1 ? 'X' : 'O');

        if (currentPlayer == 1) {
            randomMove<<<1, 1>>>(d_board, currentPlayer, time(NULL) + round);
        } else {
            smartMove<<<1, 1>>>(d_board, currentPlayer);
        }
        hipDeviceSynchronize();

        hipMemcpy(h_board, d_board, ROWS * COLS * sizeof(int), hipMemcpyDeviceToHost);
        printBoard(h_board);

        if (checkWinHost(h_board, currentPlayer)) {
            printf("Player %d (%c) wins!\n", currentPlayer, currentPlayer == 1 ? 'X' : 'O');
            break;
        }

        if (isBoardFull(h_board)) {
            printf("Game ended in a draw!\n");
            break;
        }

        currentPlayer = 3 - currentPlayer; 
        round++;
    }

    hipFree(d_board);
    free(h_board);

    printf("Game over.\n");
    return 0;
}
